/*导入主机相关库*/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <assert.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


#define total_data 512*1024*1024 //数据总量为512M
#define length_bin 256

/*用于处理内核函数调用的错误*/
__host__ void cuda_error_check(const char * prefix,const char * postfix) //前缀，后缀
{
	if(hipPeekAtLastError() != hipSuccess)
	{
		printf("\n%s%s%s",prefix,hipGetErrorString(hipGetLastError()),postfix);
		hipDeviceReset();
		//wait_exit();//等待用户的任何输入，然后退出
		exit(1);
	}
}

__shared__ unsigned int d_bin_data_shared[256];//为线程块申请一块共享内存

/*内核函数*/
__global__ void myhistogramKernel_07(
		const unsigned int * d_hist_data,
		unsigned int * d_bin_data,
		unsigned int N)
{
	/*计算出线程id*/
	const unsigned int idx = (blockIdx.x*(blockDim.x*N))+threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y)+threadIdx.y;
	const unsigned int tid = blockDim.x*N*gridDim.x*idy+idx;
	/*清理共享内存*/
	d_bin_data_shared[threadIdx.x]=0;
	/*等待线程块中所有的线程清理完共享内存*/
	__syncthreads();
	/* 读取数据并更新共享内存
	 * 每个线程一次读取32字节的数据是为了利用GPU硬件的读合并的优势，减少对全局内存的读取
	 * 一次读取N块数据是为了减少写全局内存的带宽*/
	for(unsigned int i=0,tid_offset=0;i<N;i++,tid_offset+=256)
	{
		const unsigned int value_u32 = d_hist_data[tid+tid_offset];//读取数据
		atomicAdd(&d_bin_data_shared[((value_u32 & 0xff000000)>>24)],1);//取出最高位的数据
		atomicAdd(&d_bin_data_shared[((value_u32 & 0x00ff0000)>>16)],1);//
		atomicAdd(&d_bin_data_shared[((value_u32 & 0x0000ff00)>>8)],1);
		atomicAdd(&d_bin_data_shared[value_u32 & 0x000000ff],1);//取出最后一位
	}
	/*对线程块中的所有线程进行同步，等待线程块中所有的线程全部对d_b——datain数组更新完毕*/
	__syncthreads();
	atomicAdd(&d_bin_data[threadIdx.x],d_bin_data_shared[threadIdx.x]);
}

/*主机主程序*/
int main(void)
{
	srand(time(NULL));//先种种子

	unsigned int i,s=0;
	unsigned char j;

	//用于接收cuda调用的错误码
	hipError_t err = hipSuccess;

	//输出待处理数据的长度
	printf("准备处理长为%d字节的数据\n",total_data);

	//为待处理数据分配主机端内存
	unsigned char * h_data = (unsigned char *)malloc(total_data);//malloc函数以字节为单位申请totall——data字节的内存

	//验证主机端内存分配是否成功
	if (h_data == NULL) {
	    //fprintf(stderr, "Failed to allocate host vectors!\n");
		printf("为输入数据分配主机端内存失败");
	    exit(EXIT_FAILURE);
	  }
	unsigned int *h_result = (unsigned int *)malloc(length_bin*sizeof(unsigned int));
	if (h_result == NULL) {
		//fprintf(stderr, "Failed to allocate host vectors!\n");
		printf("为输入数据分配主机端内存失败");
		exit(EXIT_FAILURE);
	}

	/*写文件*/
	/*
	fp = fopen("/home/guet-chou/eclipse-workspace/data.odt","a");//在指定目录下创建.odt文件
	for(i = 0 ; i < total_data; i ++) //产生255以内的随机数
	{
		j = rand()%255 ;
		//printf("j:%d ",j);
		fprintf(fp,"%03d",j); //把随机数写进文件
	}
	//printf("\n");
	fclose(fp); //关闭文件
	*/

	/*从文件中读取待处理数据到主内存中*/
	FILE *fp = NULL;
	unsigned int * h_data2;
	fp = fopen("/home/guet-chou/eclipse-workspace/data.odt","r");
	if(fp == NULL)
	{
	    printf("文件读取无效.\n");
	    return -1;
	}
	for(i = 0; !feof(fp); i++)
	{
		fscanf(fp, "%3d", &h_data[i]);
		if(i==536870912)
			printf("error");
	}

	fclose(fp); //关闭文件
	h_data2 = (unsigned int *)h_data;//为了充分利用GPU硬件合并读取的功能，把数据类型强制转换为整形

	/*
	 *
	 //测试数据是否正确
	unsigned int test_data;
	for(i=0;i<10;i++)
	{
		test_data = h_data2[i];
		printf("%d ",(test_data & 0xff000000)>>24);
		printf("%d ",(test_data & 0x00ff0000)>>16);
		printf("%d ",(test_data & 0x0000ff00)>>8);
		printf("%d \n",(test_data & 0x000000ff));
	}
	*/

	//在设备（即显存）中分配用于存储输入数据的内存
	unsigned int * d_data = NULL;
	err = hipMalloc((void **)&d_data,total_data);
	if (err != hipSuccess) {
	    fprintf(stderr,
	            "Failed to Call kernel functions (error code %s)!\n",
	            hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	  }
	//在设备端分配用于存储每个数出现频率的内存
	unsigned int * d_bin_data = NULL;
	err = hipMalloc((void **)&d_bin_data,length_bin*sizeof(unsigned int));
	if (err != hipSuccess) {
		    fprintf(stderr,
		            "Failed to Call kernel functions (error code %s)!\n",
		            hipGetErrorString(err));
		    exit(EXIT_FAILURE);
		  }

	//把存储在主机内存中的数据复制到设备内存中
	printf("把数据放到设备内存中\n");
	err = hipMemcpy(d_data, h_data2, total_data, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		    fprintf(stderr,
		            "Failed to Call kernel functions (error code %s)!\n",
		            hipGetErrorString(err));
		    exit(EXIT_FAILURE);
		  }

	//计算内核运行花了多少时间
	clock_t start_time, end_time;
	start_time = clock();//开始时间

	//启动内核函数
	unsigned int threadsPerBlock = 256;
	int N = 64;//每个线程块处理N个直方图
	unsigned int blocksPerGrid = (total_data/4 + threadsPerBlock - 1) / (threadsPerBlock*N);
	printf("使用每块线程块包含%d个线程的%d线程快启动内核\n",threadsPerBlock,blocksPerGrid);
	myhistogramKernel_07<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_bin_data, N);

	/*
	err = cudaGetLastError();
	if (err != cudaSuccess) {
	  fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
	          cudaGetErrorString(err));
	  exit(EXIT_FAILURE);
	}
	*/

	//主机和设备同步
	hipDeviceSynchronize();

	//结束时间
	end_time = clock();//结束时间
	printf("N=%d时花了%lf秒\n ", N,(double)(end_time - start_time) / CLOCKS_PER_SEC);//输出花了多少时间

	cuda_error_check("Error ","Returned from gmem runtime kernel");

	//把结果传回主机内存
	printf("结果传回内存\n");
	err = hipMemcpy(h_result, d_bin_data, length_bin*sizeof(unsigned int), hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
	  fprintf(stderr,
	          "结果复制到主机内存失败 (error code %s)!\n",
	          hipGetErrorString(err));
	  exit(EXIT_FAILURE);
	}

	/*打印结果*/
	for(i=0;i<10;i++)
		printf("%d:%d\n",i,h_result[i]);

	/*释放内存*/
	// Free device global memory
	err = hipFree(d_data);

	if (err != hipSuccess) {
	  fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
	          hipGetErrorString(err));
	  exit(EXIT_FAILURE);
	}

	err = hipFree(d_bin_data);

	if (err != hipSuccess) {
	  fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
	          hipGetErrorString(err));
	  exit(EXIT_FAILURE);
	}

	// Free host memory
	free(h_data);
	free(h_result);

	printf("Done\n");

	return 0;
}
